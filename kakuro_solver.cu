#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

#include <fstream>
#include <sstream>
#include <vector>

#include <bits/stdc++.h>
#include <array>

using namespace std;

enum direction{
	d_down,
	d_right,
	none
};

#define COORD std::pair<int, int>

// #define DEBUG

int iter = 0;

//////////////////////////////////////////////
// Auxiliary functions for preparing problem //
//////////////////////////////////////////////

void display_arr(int *arr, int n){

	cout << "arr: ";

	for (int i = 0; i < n; i++){
		cout << arr[i] << " ";
	}

	cout << endl;
}

void print_coords(COORD start, COORD end){
	cout << "Start:" << start.first << "," << start.second << endl;
	cout << "End:" << end.first << "," << end.second << endl;
}

int find_length(COORD start, COORD end, direction dir) {

	if (dir == d_down)
		return end.first - start.first;
	if (dir == d_right)
		return end.second - start.second;

	return -1;
}

void convert_sol(int **mat, int **&sol_mat, int m, int n){

	sol_mat = new int *[m]; // Rows
	for (int i = 0; i < m; i++){
		sol_mat[i] = new int[n]; // Cols
	}

	for (int i = 0; i < m; i++){
		for (int j = 0; j < m; j++){
			if (mat[i][j] == -2)
				sol_mat[i][j] = -2; // Empty value cell
			else
				sol_mat[i][j] = -1; // Hint or empty cell
		}
	}
}

void print_one_matrix(int **matrix, int m, int n){
	std::cout << "Matrix: " << std::endl;
	for (int i = 0; i < m; i++){ // rows
		for (int j = 0; j < n; j++){ // cols
			std::cout << matrix[i][j] << "\t";
		}
		std::cout << "\n";
	}
}

/// Auxiliary functions

struct sum{
	COORD start;
	COORD end;

	int hint;
	int dir;
	int length;
	int *arr;

	void print_sum(){
		cout << "############################" << endl;
		cout << "Creating sum with: " << endl;
		print_coords(start, end);
		cout << "Hint: " << hint << endl;
		cout << "Direction: " << dir << endl;
		cout << "Length: " << length << endl;
		cout << "############################" << endl;
	}

	sum(COORD _start, COORD _end, int _hint, direction _dir) : start(_start), end(_end), hint(_hint), dir(_dir){
		length = find_length(_start, _end, _dir);
		arr = new int[length];
#ifdef DEBUG
		cout << "############################" << endl;
		cout << "Creating sum with: " << endl;
		print_coords(start, end);
		cout << "Hint: " << hint << endl;
		cout << "Direction: " << dir << endl;
		cout << "Length: " << length << endl;
		cout << "############################" << endl;
#endif
	}

	//~sum(){
	// delete arr;
	//}
};

COORD find_end(int **matrix, int m, int n, int i, int j, direction dir){ // 0 down 1 right

	if (dir == d_right){
		for (int jj = j + 1; jj < n; jj++){
			if (matrix[i][jj] != -2 || jj == n - 1){
				if (matrix[i][jj] == -2 && jj == n - 1)
					jj++;
				COORD END = COORD(i, jj);
				return END;
			}
		}
	}

	if (dir == d_down){
		for (int ii = i + 1; ii < m; ii++){
			if (matrix[ii][j] != -2 || ii == m - 1){
				if (matrix[ii][j] == -2 && ii == m - 1)
					ii++;
				COORD END = COORD(ii, j);
				return END;
			}
		}
	}
}

vector<sum> get_sums(int **matrix, int m, int n){

	vector<sum> sums;

	for (int i = 0; i < m; i++){
		for (int j = 0; j < n; j++){
			int val = matrix[i][j];
			if (val != -1 && val != -2){
				int hint = val;
				hint = hint / 10;

				if ((hint % 100) == 0)
				{
					hint = (int)(hint / 100);
					COORD START = COORD(i, j + 1);
					COORD END = find_end(matrix, m, n, i, j, d_right);
					sum _sum = sum(START, END, hint, d_right);
					sums.push_back(_sum);
				}

				else
				{
					int div = (int)(hint / 100);
					int rem = (int)(hint % 100);

					if (div == 0 && rem != 0)
					{
						COORD START = COORD(i + 1, j);
						COORD END = find_end(matrix, m, n, i, j, d_down);
						sum _sum = sum(START, END, rem, d_down);
						sums.push_back(_sum);
					}

					if (div != 0 && rem != 0)
					{
						COORD START1 = COORD(i + 1, j);
						COORD START2 = COORD(i, j + 1);
						COORD END1 = find_end(matrix, m, n, i, j, d_down);
						COORD END2 = find_end(matrix, m, n, i, j, d_right);
						sum _sum1 = sum(START1, END1, rem, d_down);
						sum _sum2 = sum(START2, END2, div, d_right);
						sums.push_back(_sum1);
						sums.push_back(_sum2);
					}
				}
			}
		}
	}
	return sums;
}

void read_matrix(int **&matrix, std::ifstream &afile, int m, int n){

	matrix = new int *[m]; // rows

	for (int i = 0; i < m; i++){
		matrix[i] = new int[n]; // cols
	}

	int val;
	for (int i = 0; i < m; i++){
		for (int j = 0; j < n; j++){
			afile >> val;
			matrix[i][j] = val;
		}
	}
}

void sol_to_file(int **mat, int **sol_mat, int m, int n){

	string fname = "visualize.kakuro";
	ofstream to_write(fname);

	to_write << m << " " << n << "\n";

	for (int i = 0; i < m; i++){
		for (int j = 0; j < n; j++){
			if (mat[i][j] != -2)
				to_write << mat[i][j] << " ";
			else
				to_write << sol_mat[i][j] << " ";
		}
		to_write << "\n";
	}

	to_write.close();
}

//////////////////////////////////////////////
// Auxiliary functions for preparing problem //
//////////////////////////////////////////////

///////////////////////////////////////////////////
// Auxiliary functions for preparing CUDA setting //
///////////////////////////////////////////////////

void flatten_sums(vector<sum> sums, int *h_sum_starts_x, int *h_sum_starts_y, int *h_sum_ends_x, int *h_sum_ends_y, int *h_sum_hints, int *h_sum_lengths, int *h_sum_dirs, int no_sums){

	for (int i = 0; i < no_sums; i++){

		h_sum_starts_x[i] = sums[i].start.first;
		h_sum_starts_y[i] = sums[i].start.second;

		h_sum_ends_x[i] = sums[i].end.first;
		h_sum_ends_y[i] = sums[i].end.second;

		h_sum_hints[i] = sums[i].hint;
		h_sum_lengths[i] = sums[i].length;

		h_sum_dirs[i] = sums[i].dir;
	}
}

void print_flattened(int *h_sum_starts_x, int *h_sum_starts_y, int *h_sum_ends_x, int *h_sum_ends_y, int *h_sum_hints, int *h_sum_lengths, int *h_sum_dirs, int no_sums){

	cout << "###h_sum_starts_x: " << endl;
	for (int i = 0; i < no_sums; i++){
		cout << h_sum_starts_x[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_starts_y: " << endl;
	for (int i = 0; i < no_sums; i++){
		cout << h_sum_starts_y[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_ends_x: " << endl;
	for (int i = 0; i < no_sums; i++){
		cout << h_sum_ends_x[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_ends_y: " << endl;
	for (int i = 0; i < no_sums; i++){
		cout << h_sum_ends_y[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_hints: " << endl;
	for (int i = 0; i < no_sums; i++){
		cout << h_sum_hints[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_lengths: " << endl;
	for (int i = 0; i < no_sums; i++){
		cout << h_sum_lengths[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_dirs: " << endl;
	for (int i = 0; i < no_sums; i++){
		cout << h_sum_dirs[i] << " ";
	}
	cout << endl;
}

void flatten_sol_mat(int **sol_mat, int *h_sol_mat, int m, int n){
	for (int i = 0; i < m; i++){
		for (int j = 0; j < n; j++){
			h_sol_mat[i * n + j] = sol_mat[i][j];
		}
	}
}

void print_flattened_matrix(int *h_sol_mat, int m, int n){

	cout << "###Flattened matrix: " << endl;
	for (int i = 0; i < m; i++){
		for (int j = 0; j < n; j++){
			cout << h_sol_mat[i * n + j] << " ";
		}
		cout << endl;
	}
	cout << endl;
}

///////////////////////////////////////////////////
// Auxiliary functions for preparing CUDA setting //
///////////////////////////////////////////////////

///////////////////
// CUDA FUNCTIONS //
///////////////////

__global__ void kakuro_kernel(int *d_sum_starts_x, int *d_sum_starts_y, int *d_sum_ends_x, int *d_sum_ends_y,
							  int *d_sum_hints, int *d_sum_lengths, int *d_sum_dirs, int *d_sol_mat, int *d_perms, int *d_t_mats,
							  int m, int n, int no_sums, volatile bool *solved){

	// TO DO

	// About volatile bool* solved:
	// You can get idea from https://stackoverflow.com/questions/12505750/how-can-a-global-function-return-a-value-or-break-out-like-c-c-does%5B/url%5D for how to break out of a CUDA kernel
	// You may or may not use it
}

///////////////////
// CUDA FUNCTIONS //
///////////////////

int main(int argc, char **argv){

	std::string filename(argv[1]);
	std::ifstream file;
	file.open(filename.c_str());

	int m, n;
	file >> m;
	file >> n;

	int **mat;
	read_matrix(mat, file, m, n);
	print_one_matrix(mat, m, n);

	int **sol_mat;
	convert_sol(mat, sol_mat, m, n);
	// print_one_matrix(sol_mat, m, n);

	vector<sum> sums = get_sums(mat, m, n);

	// CUDA
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("==prop== Running on device: %d -- %s \n", 0, prop.name);
	printf("==prop== #of SM -- %d \n", prop.multiProcessorCount);
	printf("==prop== Max Threads Per Block: -- %d \n", prop.maxThreadsPerBlock);

	int grid_dim =		// TO DO
	int block_dim = // To DO

	int no_sums = sums.size();

	// Flattening sums and matrix
	int *h_sum_starts_x = new int[no_sums];
	int *h_sum_starts_y = new int[no_sums];
	int *h_sum_ends_x = new int[no_sums];
	int *h_sum_ends_y = new int[no_sums];
	int *h_sum_hints = new int[no_sums];
	int *h_sum_lengths = new int[no_sums];
	int *h_sum_dirs = new int[no_sums];

	flatten_sums(sums, h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, no_sums);

	print_flattened(h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, no_sums);

	int *h_sol_mat;
	h_sol_mat = new int[m * n];
	flatten_sol_mat(sol_mat, h_sol_mat, m, n);

	print_flattened_matrix(h_sol_mat, m, n);

	// Declare device pointers and copy data into device
	int *d_sum_starts_x, *d_sum_starts_y, *d_sum_ends_x, *d_sum_ends_y, *d_sum_hints, *d_sum_lengths, *d_sum_dirs, *d_sol_mat, *d_t_mats;

	hipMalloc(&d_sum_starts_x, no_sums * sizeof(int));
	hipMalloc(&d_sum_starts_y, no_sums * sizeof(int));
	hipMalloc(&d_sum_ends_x, no_sums * sizeof(int));
	hipMalloc(&d_sum_ends_y, no_sums * sizeof(int));
	hipMalloc(&d_sum_hints, no_sums * sizeof(int));
	hipMalloc(&d_sum_lengths, no_sums * sizeof(int));
	hipMalloc(&d_sum_dirs, no_sums * sizeof(int));
	hipMalloc(&d_sol_mat, (m * n) * sizeof(int));
	hipMalloc(&d_t_mats, (m * n * grid_dim * block_dim) * sizeof(int)); // Allocating invidual matrix for each GPU thread
	// You may use this array if you will implement a thread-wise solution

	hipMemcpy(d_sum_starts_x, h_sum_starts_x, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_starts_y, h_sum_starts_y, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_ends_x, h_sum_ends_x, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_ends_y, h_sum_ends_y, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_hints, h_sum_hints, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_lengths, h_sum_lengths, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_dirs, h_sum_dirs, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sol_mat, h_sol_mat, (m * n) * sizeof(int), hipMemcpyHostToDevice);

	bool *solved;
	*solved = false;
	bool *d_solved;

	hipMalloc(&d_solved, sizeof(bool));
	hipMemcpy(d_solved, solved, sizeof(bool), hipMemcpyHostToDevice);

	kakuro_kernel<<<grid_dim, block_dim>>>(d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints,
										   d_sum_lengths, d_sum_dirs, d_sol_mat, d_perms, d_t_mats, m, n,
										   no_sums, d_solved);
	hipDeviceSynchronize();
	// CUDA

	print_flattened_matrix(d_sol_mat, m, n);
	// TO DO sol_mat_flattened_to_file(mat, d_sol_mat, m, n)
	// Similiar to sol_mat, use hints from mat and values from d_sol_mat

	for (int i = 0; i < n; i++){
		delete mat[i];
		delete sol_mat[i];
	}

	delete mat;
	delete sol_mat;

	delete h_sum_starts_x;
	delete h_sum_starts_y;
	delete h_sum_ends_x;
	delete h_sum_ends_y;
	delete h_sum_hints;
	delete h_sum_lengths;
	delete h_sum_dirs;
	delete h_sol_mat;

	hipFree(d_t_mats);
	hipFree(d_sum_starts_x);
	hipFree(d_sum_starts_y);
	hipFree(d_sum_ends_x);
	hipFree(d_sum_ends_y);
	hipFree(d_sum_hints);
	hipFree(d_sum_lengths);
	hipFree(d_sum_dirs);
	hipFree(d_sol_mat);

	return 0;
}
