
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

#include <fstream>
#include <sstream>
#include <vector>

#include <bits/stdc++.h>
#include <array>
#include <omp.h>
#include <stack>

using namespace std;

int MAX_THREADS;

enum direction{
    d_down,
    d_right,
    none
};

#define COORD std::pair<int, int>

//#define DEBUG

int iter = 0;

/// Auxiliary functions

void display_arr(int *arr, int n){
    cout << "arr: ";
    for (int i = 0; i < n; i++){
        cout << arr[i] << " ";
    }
    cout << endl;
}

void print_coords(COORD start, COORD end){

    cout << "Start:" << start.first << "," << start.second << endl;
    cout << "End:" << end.first << "," << end.second << endl;
}

int find_length(COORD start, COORD end, direction dir){
    if (dir == d_down)
        return end.first - start.first;
    if (dir == d_right)
        return end.second - start.second;
    return -1;
}

void convert_sol(int **mat, int **&sol_mat, int m, int n){

    sol_mat = new int *[m]; // Rows
    for (int i = 0; i < m; i++){
        sol_mat[i] = new int[n]; // Cols
    }

    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            if (mat[i][j] == -2) 
                sol_mat[i][j] = -2; // Empty value cell
            else
                sol_mat[i][j] = -1; // Hint or empty cell
        }
    }
}

void print_one_matrix(int **matrix, int m, int n){
    std::cout << "Matrix: " << std::endl;
    for (int i = 0; i < m; i++) { // rows
        for (int j = 0; j < n; j++) { // cols
            std::cout << matrix[i][j] << "\t";
        }
        std::cout << "\n";
    }
}

void sol_to_file(int **mat, int **sol_mat, int m, int n, string fname){
    ofstream to_write(fname);

    to_write << m << " " << n << "\n";

    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            if (mat[i][j] != -2)
                to_write << mat[i][j] << " ";
            else
                to_write << sol_mat[i][j] << " ";
        }
        to_write << "\n";
    }

    to_write.close();
}

void read_matrix(int **&matrix, std::ifstream &afile, int m, int n){

    matrix = new int *[m]; // rows

    for (int i = 0; i < m; i++){
        matrix[i] = new int[n]; // cols
    }

    int val;
    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            afile >> val;
            matrix[i][j] = val;
        }
    }
}

/// Auxiliary functions

struct sum{
    COORD start;
    COORD end;

    int hint;
    int dir;
    int length;
    int posMin;
    int posMax;

    void print_sum(){
        cout << "############################" << endl;
        cout << "Creating sum with: " << endl;
        print_coords(start, end);
        cout << "Hint: " << hint << endl;
        cout << "Direction: " << dir << endl;
        cout << "Length: " << length << endl;
        cout << endl;
        cout << "############################" << endl;
    }

    sum(COORD _start, COORD _end, int _hint, direction _dir) : start(_start), end(_end), hint(_hint), dir(_dir){
        length = find_length(_start, _end, _dir);
        // This is equal to hint - sum of numbers 9 + 8 + 7
        posMin = hint - 45 + ((8-length) * (9-length)) / 2; 

        posMax = hint - (length * (length - 1)) / 2;

#ifdef DEBUG
        cout << "############################" << endl;
        cout << "Creating sum with: " << endl;
        print_coords(start, end);
        cout << "Hint: " << hint << endl;
        cout << "Direction: " << dir << endl;
        cout << "Length: " << length << endl;
        cout << "############################" << endl;
#endif
    }
    bool checkSumPartial(int** mat) const{
        int sum = 0;
        if (dir == 1){ // right
            for (int j = start.second; j < end.second; j++){
                if (mat[start.first][j] != -2){
                        sum += mat[start.first][j];
                }
                
            }
        }else{ // down
            for (int i = start.first; i < end.first; i++){
                if (mat[i][start.second] != -2){
                    sum += mat[i][start.second];
                }
            }
        }       
        
        if (sum > hint){
            return false;
        }        
        return true;
    }

    bool checkSumComplete(int ** mat) const {
        int sum = 0;
        if (dir == 1){ // right
            for (int j = start.second; j < end.second; j++){
                if (mat[start.first][j] != -2){
                        sum += mat[start.first][j];
                }
                
            }
        }else{ // down
            for (int i = start.first; i < end.first; i++){
                if (mat[i][start.second] != -2){
                    sum += mat[i][start.second];
                }
            }
        }       
        if (sum != hint){
            return false;
        }
        return true;
    }

    bool areElementsUniqueExceptEmpties(int** mat) const {
        std::unordered_set<int> unique_elements;
        if (dir == 1){ // right
            for (int j = start.second; j < end.second; j++){
                if (mat[start.first][j] != -2){
                    if (unique_elements.find(mat[start.first][j]) == unique_elements.end()){
                        unique_elements.insert(mat[start.first][j]);
                    }else{
                        // If the element is found, it means it's a duplicate, and we can return false
                        return false;
                    }  
                }
                
            }
        }
        else{ // down
            for (int i = start.first; i < end.first; i++){
                if (mat[i][start.second] != -2){
                    if (unique_elements.find(mat[i][start.second]) == unique_elements.end()){
                        unique_elements.insert(mat[i][start.second]);
                    }else{
                        return false;
                    }
                }
            }
        }       
        return true;
    }

    bool areElementsUnique(int** mat) const {
        std::unordered_set<int> unique_elements;
        if (dir == 1){ // right
            for (int j = start.second; j < end.second; j++){
                if (unique_elements.find(mat[start.first][j]) == unique_elements.end()){
                    unique_elements.insert(mat[start.first][j]);
                }else{
                    // If the element is found, it means it's a duplicate, and we can return false
                    return false;
                }  
            }
        }else{ // down
            for (int i = start.first; i < end.first; i++){
                if (unique_elements.find(mat[i][start.second]) == unique_elements.end()){
                    unique_elements.insert(mat[i][start.second]);
                }else{
                    return false;
                }
            }
        }       
        return true;
    }

    bool isArrFull(int** mat) const{
        if (dir == 1){ // right
            for (int j = start.second; j < end.second; j++){
                if (mat[start.first][j] == -2)
                    return false;
            }
        }else{ // down
            for (int i = start.first; i < end.first; i++){
                if (mat[i][start.second] == -2)
                    return false;
            }
        }       
        return true;
    }

    bool isACandidate(int val) const{
        if (val < posMin){
            return false;
        }
        if (val > posMax){
            return false;
        }
        return true;
    }

    bool fullCheck(int** mat, COORD curr, int val) const{
        int i = curr.first;
        int j = curr.second;
        if (dir == 1){ // right
            // current coordinate is not in the sum
            if (!((i == start.first) && (j >= start.second) && (j < end.second))){
                return true;
            }
        }else{ // down
            if(!(j == start.second && i >= start.first && i < end.first)){
                return true;
            }
        }
        if (!isACandidate(val)){
            return false;
        }
        if (!areElementsUniqueExceptEmpties(mat)){
            return false;
        }
        if (isArrFull(mat) || isLastCell(curr)){
            return checkSumComplete(mat) && areElementsUnique(mat);
        }else{
            return checkSumPartial(mat);
        }
    }

    bool isLastCell(COORD curr) const{
        if (dir == 1){ // to right
            if (start.first == curr.first && end.second - 1 == curr.second){
                return true;
            }
        }else{
            if (start.second == curr.second && end.first - 1 == curr.first){{
                return true;
            }}
        }
        return false;
    }
};

COORD find_end(int **matrix, int m, int n, int i, int j, direction dir){ // 0 down 1 right

    if (dir == d_right){
        for (int jj = j + 1; jj < n; jj++){
            if (matrix[i][jj] != -2 || jj == n - 1){
                if (matrix[i][jj] == -2 && jj == n - 1)
                    jj++;
                COORD END = COORD(i, jj);
                return END;
            }
        }
    }

    if (dir == d_down){
        for (int ii = i + 1; ii < m; ii++){
            if (matrix[ii][j] != -2 || ii == m - 1){
                if (matrix[ii][j] == -2 && ii == m - 1)
                    ii++;
                COORD END = COORD(ii, j);
                return END;
            }
        }
    }

    cout << "ERROR: Find end is called with faulty parameters." << endl;
    return COORD(0, 0);
}

vector<sum> get_sums(int **matrix, int m, int n){

    vector<sum> sums;

    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            int val = matrix[i][j];
            if (val != -1 && val != -2){
                int hint = val;
                hint = hint / 10;
                // right sum
                if ((hint % 100) == 0){
                    hint = (int)(hint / 100);
                    COORD START = COORD(i, j + 1);
                    COORD END = find_end(matrix, m, n, i, j, d_right);
                    sum _sum = sum(START, END, hint, d_right);
                    sums.push_back(_sum);
                }

                else{
                    int div = (int)(hint / 100);
                    int rem = (int)(hint % 100);
                    // down sum
                    if (div == 0 && rem != 0){
                        COORD START = COORD(i + 1, j);
                        COORD END = find_end(matrix, m, n, i, j, d_down);
                        sum _sum = sum(START, END, rem, d_down);
                        sums.push_back(_sum);
                    }
                    // combined sum
                    if (div != 0 && rem != 0){
                        COORD START1 = COORD(i + 1, j);
                        COORD START2 = COORD(i, j + 1);
                        COORD END1 = find_end(matrix, m, n, i, j, d_down);
                        COORD END2 = find_end(matrix, m, n, i, j, d_right);
                        sum _sum1 = sum(START1, END1, rem, d_down);
                        sum _sum2 = sum(START2, END2, div, d_right);
                        sums.push_back(_sum1);
                        sums.push_back(_sum2);
                    }
                }
            }
        }
    }
    return sums;
}

struct mat_iter{
    COORD curr;
    int **mat;
    int m, n;

    mat_iter(int **_mat, int _m, int _n) : mat(_mat), m(_m), n(_n){
        curr = COORD(0, 0);
        if (mat[curr.first][curr.second] != -2){
            set_next();
        }
    }

    // Copy constructor
    mat_iter(const mat_iter &other) {
        curr = COORD(other.curr.first, other.curr.second);
        mat = other.mat;
        m = other.m;
        n = other.n;
    }

    void set_next(){
        do {
            if (curr.second + 1 < n){
                curr = COORD(curr.first, curr.second +1);
            }else{
                curr = COORD(curr.first + 1, 0);
            }
        }while (curr.first != m && (mat[curr.first][curr.second] != -2 ));
        if (curr.first  == m ){
                curr.first = -999; // end of the iteration
                curr.second = -999;

        }
    }
};

//////////////////////////////////////////////
// Auxiliary functions for preparing problem //
//////////////////////////////////////////////

///////////////////////////////////////////////////
// Auxiliary functions for preparing CUDA setting //
///////////////////////////////////////////////////

void flatten_sums(vector<sum> sums, int *h_sum_starts_x, int *h_sum_starts_y, int *h_sum_ends_x, int *h_sum_ends_y, int *h_sum_hints, int *h_sum_lengths, int *h_sum_dirs, int no_sums){

	for (int i = 0; i < no_sums; i++){

		h_sum_starts_x[i] = sums[i].start.first;
		h_sum_starts_y[i] = sums[i].start.second;

		h_sum_ends_x[i] = sums[i].end.first;
		h_sum_ends_y[i] = sums[i].end.second;

		h_sum_hints[i] = sums[i].hint;
		h_sum_lengths[i] = sums[i].length;

		h_sum_dirs[i] = sums[i].dir;
	}
}

void print_flattened(int *h_sum_starts_x, int *h_sum_starts_y, int *h_sum_ends_x, int *h_sum_ends_y, int *h_sum_hints, int *h_sum_lengths, int *h_sum_dirs, int no_sums){

	cout << "###h_sum_starts_x: " << endl;
	for (int i = 0; i < no_sums; i++){
		cout << h_sum_starts_x[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_starts_y: " << endl;
	for (int i = 0; i < no_sums; i++){
		cout << h_sum_starts_y[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_ends_x: " << endl;
	for (int i = 0; i < no_sums; i++){
		cout << h_sum_ends_x[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_ends_y: " << endl;
	for (int i = 0; i < no_sums; i++){
		cout << h_sum_ends_y[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_hints: " << endl;
	for (int i = 0; i < no_sums; i++){
		cout << h_sum_hints[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_lengths: " << endl;
	for (int i = 0; i < no_sums; i++){
		cout << h_sum_lengths[i] << " ";
	}
	cout << endl;

	cout << "###h_sum_dirs: " << endl;
	for (int i = 0; i < no_sums; i++){
		cout << h_sum_dirs[i] << " ";
	}
	cout << endl;
}

void flatten_sol_mat(int **sol_mat, int *h_sol_mat, int m, int n){
	for (int i = 0; i < m; i++){
		for (int j = 0; j < n; j++){
			h_sol_mat[i * n + j] = sol_mat[i][j];
		}
	}
}

void print_flattened_matrix(int *h_sol_mat, int m, int n){

	cout << "###Flattened matrix: " << endl;
	for (int i = 0; i < m; i++){
		for (int j = 0; j < n; j++){
			cout << h_sol_mat[i * n + j] << " ";
		}
		cout << endl;
	}
	cout << endl;
}

///////////////////////////////////////////////////
// Auxiliary functions for preparing CUDA setting //
///////////////////////////////////////////////////
struct state{
    mat_iter iter;
    int val;
    state(mat_iter _iter, int _val): iter(_iter), val(_val){}
};
///////////////////
// CUDA FUNCTIONS //
///////////////////

__global__ 
    void kakuro_kernel(int *d_sum_starts_x, int *d_sum_starts_y, int *d_sum_ends_x, int *d_sum_ends_y, int *d_sum_hints, 
    int *d_sum_lengths, int *d_sum_dirs, int *d_sol_mat, int *d_t_mats, int m, int n, int no_sums,  volatile bool *solved){
        *solved = true;
            

	// TO DO

	// About volatile bool* solved:
	// You can get idea from https://stackoverflow.com/questions/12505750/how-can-a-global-function-return-a-value-or-break-out-like-c-c-does%5B/url%5D for how to break out of a CUDA kernel
	// You may or may not use it
}

///////////////////
// CUDA FUNCTIONS //
///////////////////



bool solution(int **sol_mat, vector<sum> &sums, int m, int n, stack<state> state_stack) {    
    while(!state_stack.empty()){
        mat_iter iter = state_stack.top().iter;
        int curr_val = state_stack.top().val;
        state_stack.pop();
        if (curr_val < 10){
        
            // Update the matrix with the new val for iter.curr
            sol_mat[iter.curr.first][iter.curr.second] = curr_val;
            bool partial_correctness = true;
            for (int i = 0; i< sums.size(); i++){
                if (!sums[i].fullCheck(sol_mat, iter.curr, curr_val)){
                    partial_correctness = false;
                    break;
                }
            }
            if (partial_correctness){
                state_stack.push(state(iter, curr_val));
                mat_iter temp(iter);
                temp.set_next();
                if (temp.curr.first == -999 || temp.curr.second == -999){
                    cout << "END INSIDE:" << endl;
                    print_one_matrix(sol_mat, m, n);
                    return true;
                }
                state_stack.push(state(temp, 1));
            }else{
                curr_val += 1;
                sol_mat[iter.curr.first][iter.curr.second] = -2;
                state_stack.push(state(iter, curr_val));
            }
                
        }else{
            while(!state_stack.empty()){
                mat_iter iter = state_stack.top().iter;
                int curr_val = state_stack.top().val;
                state_stack.pop();
                sol_mat[iter.curr.first][iter.curr.second] = -2;
                curr_val += 1;
                if (curr_val < 10){
                    state_stack.push(state(iter, curr_val));
                    break;
                }
            }
        }
    }
    return false;
}


int main(int argc, char **argv){

    std::string filename(argv[1]);
    std::ifstream file;
    file.open(filename.c_str());

    int m, n;
    
    file >> m;
    file >> n;

    int **mat;
    read_matrix(mat, file, m, n);
    print_one_matrix(mat, m, n);

    int **sol_mat;
    convert_sol(mat, sol_mat, m, n);
    print_one_matrix(sol_mat, m, n);

    double start;
    double end;

    // CUDA
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("==prop== Running on device: %d -- %s \n", 0, prop.name);
	printf("==prop== #of SM -- %d \n", prop.multiProcessorCount);
	printf("==prop== Max Threads Per Block: -- %d \n", prop.maxThreadsPerBlock);

    vector<sum> sums = get_sums(mat, m, n);
    
    int grid_dim =  m;		// TO DO
	int block_dim = n;// To DO

    int no_sums = sums.size();

	// Flattening sums and matrix
	int *h_sum_starts_x = new int[no_sums];
	int *h_sum_starts_y = new int[no_sums];
	int *h_sum_ends_x = new int[no_sums];
	int *h_sum_ends_y = new int[no_sums];
	int *h_sum_hints = new int[no_sums];
	int *h_sum_lengths = new int[no_sums];
	int *h_sum_dirs = new int[no_sums];


    flatten_sums(sums, h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, no_sums);

	print_flattened(h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, no_sums);

    int *h_sol_mat;
	h_sol_mat = new int[m * n];
	flatten_sol_mat(sol_mat, h_sol_mat, m, n);

	print_flattened_matrix(h_sol_mat, m, n);

    // Declare device pointers and copy data into device
	int *d_sum_starts_x, *d_sum_starts_y, *d_sum_ends_x, *d_sum_ends_y, *d_sum_hints, *d_sum_lengths, *d_sum_dirs, *d_sol_mat, *d_t_mats;

	hipMalloc(&d_sum_starts_x, no_sums * sizeof(int));
	hipMalloc(&d_sum_starts_y, no_sums * sizeof(int));
	hipMalloc(&d_sum_ends_x, no_sums * sizeof(int));
	hipMalloc(&d_sum_ends_y, no_sums * sizeof(int));
	hipMalloc(&d_sum_hints, no_sums * sizeof(int));
	hipMalloc(&d_sum_lengths, no_sums * sizeof(int));
	hipMalloc(&d_sum_dirs, no_sums * sizeof(int));
	hipMalloc(&d_sol_mat, (m * n) * sizeof(int));
	hipMalloc(&d_t_mats, (m * n * grid_dim * block_dim) * sizeof(int)); // Allocating invidual matrix for each GPU thread
	// You may use this array if you will implement a thread-wise solution

    hipMemcpy(d_sum_starts_x, h_sum_starts_x, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_starts_y, h_sum_starts_y, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_ends_x, h_sum_ends_x, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_ends_y, h_sum_ends_y, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_hints, h_sum_hints, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_lengths, h_sum_lengths, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum_dirs, h_sum_dirs, no_sums * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sol_mat, h_sol_mat, (m * n) * sizeof(int), hipMemcpyHostToDevice);

    bool *solved = (bool*)malloc(sizeof(bool));
	*solved = false;
	bool *d_solved;

	hipMalloc(&d_solved, sizeof(bool));
	hipMemcpy(d_solved, solved, sizeof(bool), hipMemcpyHostToDevice);
    
    mat_iter iter = mat_iter(sol_mat,  m, n);
    stack<state> state_stack;
    state_stack.push(state(iter, 1));
    cout << "Number of threads: " << omp_get_num_threads() << endl;
    
    start = omp_get_wtime();
    kakuro_kernel<<<grid_dim, block_dim>>>(d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints,
										   d_sum_lengths, d_sum_dirs, d_sol_mat, d_t_mats, m, n,
										   no_sums, d_solved);
	hipDeviceSynchronize();
    bool result = solution(sol_mat, sums, m, n, state_stack);
    end = omp_get_wtime(); 
    printf("Work took %f seconds\n", end - start);

    sol_to_file(mat, sol_mat, m, n, "solution.kakuro");
    if (result){
        cout << "SUCCESS" << endl;
    }else{
        cout << "COULD NOT SOLVE" << endl;
    }
    
    cout << "SOLVED BY CUDA before copy: " << *solved << endl; 

    hipMemcpy(solved, d_solved, sizeof(bool), hipMemcpyDeviceToHost);
    cout << "SOLVED BY CUDA?: " << *solved << endl; 
    


    // DELETE PART 
    for (int i = 0; i < m; i++){
        delete mat[i];
        delete sol_mat[i];
    }

    delete mat;
    delete sol_mat;

	delete h_sum_starts_x;
	delete h_sum_starts_y;
	delete h_sum_ends_x;
	delete h_sum_ends_y;
	delete h_sum_hints;
	delete h_sum_lengths;
	delete h_sum_dirs;
	delete h_sol_mat;

	hipFree(d_t_mats);
	hipFree(d_sum_starts_x);
	hipFree(d_sum_starts_y);
	hipFree(d_sum_ends_x);
	hipFree(d_sum_ends_y);
	hipFree(d_sum_hints);
	hipFree(d_sum_lengths);
	hipFree(d_sum_dirs);
	hipFree(d_sol_mat);

    return 0;
}
